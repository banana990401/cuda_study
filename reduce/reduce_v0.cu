#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


// v0: naive版本
// latency: 0.959590 ms
// blockSize作为模板参数的效果主要用于静态shared memory的申请需要传入编译期常量指定大小（L10)
template <int blockSize>
__global__ void reduce_v0(float* d_in, float* d_out)
{
    int tid   = threadIdx.x;
    int g_tid = blockIdx.x * blockSize + threadIdx.x;

    __shared__ float smem[blockSize];

    smem[tid] = d_in[g_tid];
    __syncthreads();

    for(int stride = 1; stride < blockSize; stride *= 2)
    {
        if((tid & (2 * stride - 1)) == 0)
        {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }

    if(tid == 0)
    {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float* out, float groudtruth, int n)
{
    float res = 0;
    for(int i = 0; i < n; i++)
    {
        res += out[i];
    }
    if(res != groudtruth)
    {
        return false;
    }
    return true;
}

int main()
{
    float milliseconds = 0;
    const int N        = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize        = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    // int GridSize = 100000;
    float* a = (float*)malloc(N * sizeof(float));
    float* d_a;
    hipMalloc((void**)&d_a, N * sizeof(float));

    float* out = (float*)malloc((GridSize) * sizeof(float));
    float* d_out;
    hipMalloc((void**)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++)
    {
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    // warm up
    reduce_v0<blockSize><<<Grid, Block>>>(d_a, d_out);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    int iter = 10;
    for(int i = 0; i < iter; i++)
    {
        reduce_v0<blockSize><<<Grid, Block>>>(d_a, d_out);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d \n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right)
    {
        printf("the ans is right\n");
    }
    else
    {
        printf("the ans is wrong\n");
        // for(int i = 0; i < GridSize;i++){
        // printf("res per block : %lf ",out[i]);
        //}
        // printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v0 latency = %f ms\n", milliseconds / iter);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}
