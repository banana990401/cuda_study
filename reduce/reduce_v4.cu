#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


// v4: 循环展开
// latency: 0.414413 ms
#define THREAD_PER_BLOCK 256

template <int blockSize>
__device__ void blockReduce(float* smem)
{
    if(blockSize >= 1024)
    {
        if(threadIdx.x < 512)
        {
            smem[threadIdx.x] += smem[threadIdx.x + 512];
        }
        __syncthreads();
    }
    if(blockSize >= 512)
    {
        if(threadIdx.x < 256)
        {
            smem[threadIdx.x] += smem[threadIdx.x + 256];
        }
        __syncthreads();
    }
    if(blockSize >= 256)
    {
        if(threadIdx.x < 128)
        {
            smem[threadIdx.x] += smem[threadIdx.x + 128];
        }
        __syncthreads();
    }
    if(blockSize >= 128)
    {
        if(threadIdx.x < 64)
        {
            smem[threadIdx.x] += smem[threadIdx.x + 64];
        }
        __syncthreads();
    }
    // the final warp
    if(threadIdx.x < 32)
    {
        volatile float* vshm = smem;

        vshm[threadIdx.x] += vshm[threadIdx.x + 32];
        vshm[threadIdx.x] += vshm[threadIdx.x + 16];
        vshm[threadIdx.x] += vshm[threadIdx.x + 8];
        vshm[threadIdx.x] += vshm[threadIdx.x + 4];
        vshm[threadIdx.x] += vshm[threadIdx.x + 2];
        vshm[threadIdx.x] += vshm[threadIdx.x + 1];
    }
}

template <int blockSize>
__global__ void reduce_v3(float* d_in, float* d_out)
{
    int tid   = threadIdx.x;
    int g_tid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    __shared__ float smem[THREAD_PER_BLOCK];

    smem[tid] = d_in[g_tid] + d_in[g_tid + blockDim.x];
    __syncthreads();

    blockReduce<blockSize>(smem);

    if(tid == 0)
    {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float* out, float groudtruth, int n)
{
    float res = 0;
    for(int i = 0; i < n; i++)
    {
        res += out[i];
    }
    if(res != groudtruth)
    {
        return false;
    }
    return true;
}

int main()
{
    float milliseconds = 0;
    const int N        = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize        = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    // int GridSize = 100000;
    float* a = (float*)malloc(N * sizeof(float));
    float* d_a;
    hipMalloc((void**)&d_a, N * sizeof(float));

    float* out = (float*)malloc((GridSize) * sizeof(float));
    float* d_out;
    hipMalloc((void**)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++)
    {
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize / 2);

    // warm up
    reduce_v3<blockSize / 2><<<Grid, Block>>>(d_a, d_out);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    int iter = 10;
    for(int i = 0; i < iter; i++)
    {
        reduce_v3<blockSize / 2><<<Grid, Block>>>(d_a, d_out);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d \n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right)
    {
        printf("the ans is right\n");
    }
    else
    {
        printf("the ans is wrong\n");
        // for(int i = 0; i < GridSize;i++){
        // printf("res per block : %lf ",out[i]);
        //}
        // printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v3 latency = %f ms\n", milliseconds / iter);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}
