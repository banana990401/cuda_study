#include <hip/hip_runtime.h>
#include <bits/stdc++.h>


#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// biasadd + mask + scale + elemwise_add四个算子的融合
// （x + bias） * mask * scale + addend;

template <typename T>
struct MaskScaleAndElemwiseAddFunctor
{
    MaskScaleAndElemwiseAddFunctor(const uint8_t* mask, const T* add_val, float scale)
        : _mask(mask), _add_val(add_val), _scale(scale)
    {
    }

    __device__ T operator()(T x, int i) const
    {
        return x * static_cast<T>(static_cast<bool>(_mask[i]) * _scale) + _add_val[i];
    }

    const uint8_t* _mask;
    const T* _add_val;
    float _scale;
};

// naive
template <int biasSize, typename FUNCTOR, typename T>
__global__ void
FusedBaisAdd(FUNCTOR functor, T* dx, T* dy, T* d_bias, const int n, const int bias_size)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    for(int i = gid; i < n; i += blockDim.x * gridDim.x)
    {
        T tmp = dx[i] + d_bias[i % bias_size];
        dy[i] = functor(tmp, i);
    }
}

// 向量化访存
template <int biasSize, typename FUNCTOR, typename T>
__global__ void
FusedBaisAddVecSmem(FUNCTOR functor, T* dx, T* dy, T* d_bias, const int n, const int bias_size)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    __shared__ T smem[biasSize];

    if(tid < biasSize)
        smem[tid] = d_bias[tid];
    __syncthreads();

    for(int i = gid; i < n / 4; i += blockDim.x * gridDim.x)
    {
        float4 a = reinterpret_cast<float4*>(dx)[i];
        float4 b;

        b.x = functor(a.x + smem[(i * 4) % biasSize], i * 4);
        b.y = functor(a.y + smem[(i * 4 + 1) % biasSize], i * 4 + 1);
        b.z = functor(a.z + smem[(i * 4 + 2) % biasSize], i * 4 + 2);
        b.w = functor(a.w + smem[(i * 4 + 3) % biasSize], i * 4 + 3);

        reinterpret_cast<float4*>(dy)[i] = b;
    }
}

bool CheckRight(float* y, float* groudTruth, const int n)
{
    for(int i = 0; i < n; ++i)
    {
        if(y[i] != groudTruth[i])
        {
            printf("y[%d] : %f \n", i, y[i]);
            printf("groundTruth[%d] : %f\n", i, groudTruth[i]);
            return false;
        }
    }
    return true;
}

int main()
{
    constexpr int n         = 100000;
    constexpr int bias_size = 10;

    float scale          = 0.5;
    uint8_t* mask_tensor = new uint8_t[n];
    float* add_val       = new float[n];
    // 初始化
    for(int i = 0; i < n; ++i)
    {
        mask_tensor[i] = (uint8_t)(i);
        add_val[i]     = (float)(i);
    }

    float* x    = (float*)malloc(sizeof(float) * n);
    float* y    = (float*)malloc(sizeof(float) * n);
    float* bias = (float*)malloc(sizeof(float) * bias_size);
    for(int i = 0; i < n; ++i)
    {
        x[i] = (float)(i);
        y[i] = 0.0f;
    }
    for(int i = 0; i < bias_size; ++i)
        bias[i] = i;

    float* groudTruth = (float*)malloc(sizeof(float) * n);
    for(int i = 0; i < n; ++i)
    {
        groudTruth[i] = (x[i] + bias[i % bias_size]) *
                            static_cast<float>(static_cast<bool>(mask_tensor[i]) * scale) +
                        add_val[i];
    }

    float *dx, *dy, *d_bias;
    hipMalloc((void**)&dx, sizeof(float) * n);
    hipMalloc((void**)&dy, sizeof(float) * n);
    hipMalloc((void**)&d_bias, sizeof(float) * bias_size);
    hipMemcpy(dx, x, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(dy, y, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, sizeof(float) * bias_size, hipMemcpyHostToDevice);
    uint8_t* d_mask_tensor;
    float* d_add_val;
    hipMalloc((void**)&d_mask_tensor, sizeof(uint8_t) * n);
    hipMalloc((void**)&d_add_val, sizeof(float) * n);
    hipMemcpy(d_mask_tensor, mask_tensor, sizeof(uint8_t) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_add_val, add_val, sizeof(float) * n, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int blockSize = 512;
    int gridSize  = std::min((n + blockSize - 1) / blockSize, deviceProp.maxGridSize[0]);

    MaskScaleAndElemwiseAddFunctor<float> functor(d_mask_tensor, d_add_val, scale);

    dim3 Block(blockSize);
    dim3 Grid(gridSize);

    float milliseconds = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for(int i = 0; i < 1000; ++i)
        FusedBaisAddVecSmem<bias_size><<<Grid, Block>>>(functor, dx, dy, d_bias, n, bias_size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(y, dy, sizeof(float) * n, hipMemcpyDeviceToHost);

    bool isRight = CheckRight(y, groudTruth, n);
    if(isRight)
        printf("结果正确\n");
    else
        printf("结果错误\n");

    printf("it costs %f s \n", milliseconds / 1000);

    hipFree(dx);
    hipFree(dy);
    hipFree(d_bias);
    hipFree(d_add_val);
    hipFree(d_mask_tensor);
    free(x);
    free(y);
    free(bias);
    free(groudTruth);
    delete mask_tensor;
    mask_tensor = nullptr;
    delete add_val;
    add_val = nullptr;
}